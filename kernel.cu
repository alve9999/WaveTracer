#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <thrust/complex.h>
#include <iostream>
#include "defines.h"

static thrust::complex<double>* intensity;
static uint8_t* image;
__global__ void initializeVariables(thrust::complex<double>* intensity, uint8_t* image) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = i * ny + j;
    intensity[idx] = 0;
    image[idx * 4]=0;
    image[idx * 4+1]=0;
    image[idx * 4+2]=0;
    image[idx * 4+3]=255;
}

void CUDA_INIT() {
	hipMalloc(&intensity,nx * ny * sizeof(thrust::complex<double>));
    hipMalloc(&image, 4 * nx * ny * sizeof(uint8_t));
    dim3 dimGrid(nx / BLOCK_SIZE, ny / BLOCK_SIZE, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    initializeVariables << <dimGrid, dimBlock >> > (intensity, image);
}

__global__ void dev_apply_light(double x, double y,thrust::complex<double>* intensity){
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    const thrust::complex<double> k(0.0, 1.0);
    double d = std::pow(((double)y)-((double)i * outscale ), 2) + std::pow(((double)x)-((double)j * outscale), 2);
    double theta = std::atan(std::sqrt(d) / L);
    double skew = (1 + std::cos(theta)) / 2;
    double r = std::sqrt(d + std::pow(L, 2));
    double phase = 2.0 * 3.14159265 * std::sqrt(d+std::pow(L,2)) / wavelenght;
    thrust::complex<double> res = (I_0 * thrust::exp(k * phase)) * skew * L / (d + std::pow(L, 2));
    intensity[i * ny + j] += (I_0 * thrust::exp(k * phase)) * skew *  L / (d + std::pow(L, 2));
}

void apply_light(double x, double y) {
    dim3 dimGrid(nx / BLOCK_SIZE, ny / BLOCK_SIZE, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dev_apply_light<<<dimGrid,dimBlock>>>(x, y, intensity);
}
__device__ uint8_t dev_get_colour(int i, int j, thrust::complex<double>* intensity) {
    int idx = i * ny + j;
    double val = thrust::norm(intensity[idx]);
    if (val > 255) {
        return 255;
    }
    else {
        
        return (uint8_t)(val);
    }
}
__global__ void kernel_create_color(thrust::complex<double>* intensity, uint8_t* image) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    image[(i * ny + j) * 4] = dev_get_colour(i,j,intensity);
}
void get_colour(uint8_t* pixel_buffer) {
    dim3 dimGrid(nx / BLOCK_SIZE, ny / BLOCK_SIZE, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    check_error();
    kernel_create_color << <dimGrid, dimBlock >> > (intensity, image);
    check_error();
    hipMemcpy(pixel_buffer, image, 4 * ny * nx * sizeof(uint8_t), hipMemcpyDeviceToHost);
    check_error();
}